#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include "time_invocation.hpp"

template<class T>
double time(size_t n)
{
  auto unsorted = random_device_vector<T>(n);
  thrust::device_vector<T> vec(n);

  auto reset = [&]() mutable
  {
    vec = unsorted;
  };

  auto time_me = [&]
  {
    thrust::stable_sort(vec.begin(), vec.end());
    hipDeviceSynchronize();
  };

  size_t us = time_invocation_with_reset_in_microseconds(100, time_me, reset);

  return static_cast<double>(us) / 1000000;
}

int main(int argc, char** argv)
{
  double (*call_me)(size_t) = time<int>;
  std::string type = "int";

  if(argc >= 2)
  {
    type = argv[1];
  }

  size_t n = 1 << 20;

  if(argc >= 3)
  {
    n = atoi(argv[2]);
  }

  if(type == "int")
  {
    call_me = time<int>;
  }
  else if(type == "long")
  {
    call_me = time<uint64_t>;
  }
  else if(type == "float")
  {
    call_me = time<float>;
  }
  else if(type == "double")
  {
    call_me = time<double>;
  }
  else
  {
    throw std::runtime_error("Unrecognized type");
  }

  double seconds = call_me(n);

  std::cout << n << ", " << seconds;

  return 0;
}


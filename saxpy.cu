#include "time_invocation.hpp"
#include <thrust/device_vector.h>
#include <thrust/transform.h>


template<class T>
struct saxpy_functor
{
  T a;

  __host__ __device__
  T operator()(const T& x, const T& y) const
  {
    return a * x + y;
  }
};


template<class T>
double time(size_t n)
{
  thrust::device_vector<T> vec1(n);
  thrust::device_vector<T> vec2(n);
  thrust::device_vector<T> vec3(n);

  auto us = time_invocation_in_microseconds(100, [&]
  {
    thrust::transform(vec1.begin(), vec1.end(), vec2.begin(), vec3.begin(), saxpy_functor<T>(13));

    hipDeviceSynchronize();
  });

  return static_cast<double>(us) / 1000000;
}


int main(int argc, char** argv)
{
  double (*call_me)(size_t) = time<int>;
  std::string type = "int";

  if(argc >= 2)
  {
    type = argv[1];
  }

  size_t n = 1 << 20;

  if(argc >= 3)
  {
    n = atoi(argv[2]);
  }

  if(type == "int")
  {
    call_me = time<int>;
  }
  else if(type == "long")
  {
    call_me = time<uint64_t>;
  }
  else if(type == "float")
  {
    call_me = time<float>;
  }
  else if(type == "double")
  {
    call_me = time<double>;
  }
  else
  {
    throw std::runtime_error("Unrecognized type");
  }

  std::clog << "T: " << type << std::endl;
  std::clog << "n: " << n << std::endl;

  double seconds = call_me(n);

  std::clog << "s: " << seconds << std::endl;

  std::cout << seconds;

  return 0;
}

